// Easy

#include <hip/hip_runtime.h>

__global__ void count_equal_kernel(const int* input, int* output, int N, int K) {
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const int* input, int* output, int N, int K) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    count_equal_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, K);
    hipDeviceSynchronize();
}