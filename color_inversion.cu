// Easy

#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int pixel_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    
    // Check if the thread is within the valid pixel range
    if (pixel_idx < total_pixels) {
        // Each pixel has 4 components (RGBA)
        int base_idx = pixel_idx * 4;
        // Invert R, G, B; leave A unchanged
        image[base_idx + 0] = 255 - image[base_idx + 0]; // R
        image[base_idx + 1] = 255 - image[base_idx + 1]; // G
        image[base_idx + 2] = 255 - image[base_idx + 2]; // B
        // image[base_idx + 3] remains unchanged (A)
    }
}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}