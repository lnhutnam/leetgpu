// Easy

#include <hip/hip_runtime.h>

__global__ void silu_kernel(const float* input, float* output, int N) {

}

// input, output are device pointers
extern "C" void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    silu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}

