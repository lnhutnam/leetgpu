// Easy

#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {

}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}