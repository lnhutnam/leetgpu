// Easy

#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {

}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}