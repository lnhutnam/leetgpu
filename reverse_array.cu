// Easy

#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {

}

// input is device pointer
extern "C" void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}