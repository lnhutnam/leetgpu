// Easy

#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {

}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}